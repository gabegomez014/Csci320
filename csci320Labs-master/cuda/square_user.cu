
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float * d_out, float * d_in){
    int idx = blockIdx.x *blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int argc, char ** argv) {
	
	const int ARRAY_SIZE=atoi(argv[1]);
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	//float h_in[ARRAY_SIZE];
	float *h_in;
	float *h_out;
	//cudaMalloc((void**) &h_in, ARRAY_BYTES);
	//cudaMalloc((void**) &h_out, ARRAY_BYTES);
	h_in = (float *) malloc(ARRAY_BYTES);
	h_out = (float *) malloc(ARRAY_BYTES);
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	//float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	int blocks = 1;
	for (int k = 0; k < ARRAY_SIZE; k++) {
		if (ARRAY_SIZE / blocks <= 1024) {
			break;
		}
	
		else { blocks += 1; }
	}
	square<<<blocks,ARRAY_SIZE/blocks>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipError_t err;
	
	if (hipSuccess != (err = hipGetLastError()) ) {
		fprintf(stderr, "CUDA errror: %s\n", hipGetErrorString(err));
		exit(-2);
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
